#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <string.h>
#include <time.h>
#include <math.h>
#include <hip/hip_runtime.h>

#include <hiprand/hiprand_kernel.h>

#define NTHREADS_PER_BLOCK 256
#define BLOCKS 64

float inverse_gausian(float median,float sigma,float random_value){
  float y1=median + sqrt(-2*pow(sigma,2)*log(sqrt(2*M_PI)*sigma*random_value));
  float y2=median - sqrt(-2*pow(sigma,2)*log(sqrt(2*M_PI)*sigma*random_value));
  float x=(float)rand()/RAND_MAX;
  return (x<0.5)?y1:y2;
}

void apply_Noise(int rows,int cols,float** In,float** Out,float sigma){
  for (int i=0; i<rows; i++)
    for (int j=0; j<cols; j++){
      double z=inverse_gausian(In[i][j],sigma,(float)rand()/RAND_MAX);
      if (z > 1)
        Out[i][j]=1;
      else if (z < 0)
        Out[i][j]=0;
      else
        Out[i][j]=z;
    }
}

__global__
void d_fill(float* B,float* d,float* patches,int i,int patchsize,float* gauss,int n,int size_patches,float filtsigma){

  // each block has i.e. p^2 elements starting from the top row
  // therefore number of cols of specific block is n
  // and blockdim = rows*n ==> rows=(blockdim/n)
  // filling the array of patches is like a 2d stencil problem
  // passing the patches of one block in a shared memory
  int h_patch=(patchsize-1)/2;
  int size_rows=n+2*h_patch; // basically number of columns
  extern __shared__ float sh_patches[];
  int index = blockIdx.x*blockDim.x+threadIdx.x;

  // adding h_patch in order to navigate through the B array
  // basically copying a part of B array to shared memory
  int x=index/n+h_patch;
  int y=index%n+h_patch;
  int l_x=threadIdx.x/n;
  int l_y=threadIdx.x%n;
  if (index<(n*n)){
    sh_patches[(l_x+h_patch)*size_rows+(l_y+h_patch)]=B[x*size_rows+y];
    if ((l_y<h_patch)&&(l_x<h_patch)){
      sh_patches[l_x*size_rows+l_y]=B[(x-h_patch)*size_rows+(y-h_patch)];
      sh_patches[(l_x+h_patch+blockDim.x/n)*size_rows+(l_y+h_patch+n)]=B[(x+n)*size_rows+(y+n)];
    }
    else if (l_y<h_patch){
      sh_patches[(l_x+h_patch)*size_rows+l_y]=B[x*size_rows+(y-h_patch)];
      sh_patches[(l_x+h_patch)*size_rows+(l_y+h_patch+n)]=B[x*size_rows+(y+n)];
    }
    else if ((l_x<h_patch)){
      sh_patches[(l_x)*size_rows+(l_y+h_patch)]=B[(x-h_patch)*size_rows+y];
      sh_patches[(l_x+h_patch+blockDim.x/n)*size_rows+(l_y+h_patch)]=B[(x+n)*size_rows+y];
    }
  }

  __syncthreads();

  // d_filling the array by taking advantage the shared memory above
  // each element is (x+blockIdx.x),(y+blockIdx.x)
  // if i belongs to the same block as our thread then both readings
  // come from shared memory
  // else the reading of element i comes from global


  int f_index=(l_x)*n*patchsize*patchsize+(l_y)*patchsize*patchsize;
  // normilizing our data
  x=i/n;
  y=i%n;
  float sum=0;
  // in order to check whether the element comes from the same block
  if ((i/blockDim.x)==(blockIdx.x)){
    int n_i=i%blockDim.x;
    x=n_i/n;
    y=n_i%n;
    int s_index=x*n*patchsize*patchsize+y*patchsize*patchsize;
    if (index < powf(n,2)){
      if (index==n_i)
        d[index*n*n+i]=1;
      else {
        for (int j=0; j<patchsize; j++)
          for (int k=0; k<patchsize; k++)
            sum+=gauss[j]*powf(sh_patches[f_index+k*patchsize+j]-sh_patches[s_index+k*patchsize+j],2);
        d[index*n*n+i]=expf(-sum/powf(filtsigma,2));
        }
      }
    }
    if (index==i)
      d[index*n*n+i]=1;
    else{
      x=i/n;
      y=i%n;
      int s_index=x*n*patchsize*patchsize+y*patchsize*patchsize;
      for (int j=0; j<patchsize; j++)
        for (int k=0; k<patchsize; k++)
          sum+=gauss[j]*powf(sh_patches[f_index+k*patchsize+j]-patches[s_index+k*patchsize+j],2);
      d[index*n*n+i]=expf(-sum/powf(filtsigma,2));
    }
}

// get the patch of an image in points (x,y)
// with patchsize
__global__
void patch(float* B,float* patches,int i,int j,int size,int patchsize,int n){
  int index = blockIdx.x * NTHREADS_PER_BLOCK + threadIdx.x;
  int h_patch=(patchsize-1)/2;
  int x=index/size;
  int y=index%size;
    if ((x<size)&&(y<size)){
        if ((y>=h_patch)&&(y<n+h_patch)&&(x<n+h_patch)&&(x>=h_patch)){
            int patch_index=(x-h_patch)*n*patchsize*patchsize+(y-h_patch)*patchsize*patchsize+j*patchsize+i;
            patches[patch_index]=B[(x+i-h_patch)*size+(y+j-h_patch)];
    }
  }
}


// function to mirror the array
__global__
void mirroring(float* A,float* B,int patchsize,int n){
  int h_patch=(patchsize-1)/2;
  int size=n+2*h_patch;
  int index = blockIdx.x * NTHREADS_PER_BLOCK + threadIdx.x;
  int x= index/size; // row
  int y= index%size; // col
  if (index<(size*size))
    if ((x<h_patch)||(x>=n+h_patch)||(y<h_patch)||(y>=n+h_patch))
      B[index]=0;
    else
      B[index]=A[(x-h_patch)*n+(y-h_patch)];
}

__global__
void matrix_vec(float* d,float* vec,float* out,int n){
  int index= blockIdx.x*blockDim.x+threadIdx.x;
  int x=index/n;
  int y=index%n;
  int d_x=x*powf(n,3);
  int d_y=y*powf(n,2);
  float sum=0;
  float sum1=0;
  if (index<(n*n)){
    for(int i=0; i<(n*n); i++){
      sum+=d[d_x+d_y+i]*vec[i];
      sum1+=d[d_x+d_y+i];
    }
    out[index]=sum/sum1;
  }
}

void filter(float** A,int n,int patchsize,float patchsigma,float filtsigma){


  int h_patch=(patchsize-1)/2;
  int size=n+2*h_patch;

  // generating image as 1d array
  float* temp_out=(float*)malloc(n*n*sizeof(float));
  for(int i=0; i<n; i++)
    for (int j=0; j<n; j++)
      temp_out[i*n+j]=A[i][j];

  // calculate gaussian weight
  // calculate distance for each patch
  float* gauss=(float*)malloc(patchsize*patchsize*sizeof(float));
  float sum=0;
  for (int i=0; i<patchsize*patchsize; i++){
    int x=i/(patchsize);
    int y=i%(patchsize);
    x=x-((patchsize)-1)/2;
    y=y-((patchsize)-1)/2;
    gauss[i]= (1/(2.0*patchsigma))*exp(-(x*x+y*y)/(2*M_PI*patchsigma));
    sum+=(1/(2.0*patchsigma))*exp(-(x*x+y*y)/(2*M_PI*patchsigma));
  }
  for (int i=0; i<patchsize*patchsize; i++)
    gauss[i]/=sum;

  // CUDA mirroring, the allocation of variables dev is not being done
  // immediately
  float *dev_b,*dev_a;
  hipMalloc((void**)&dev_a,n*n*sizeof(float));
  hipMalloc((void**)&dev_b,size*size*sizeof(float));
  hipMemcpy(dev_a,temp_out,n*n*sizeof(float),hipMemcpyHostToDevice);
  // size*size+NTHREADS_PER_BLOCK-1/NTHREADS_PER_BLOCK because if
  // size*size is < NTHREADS_PER_BLOCK integer division will give 0
  mirroring<<<(size*size+NTHREADS_PER_BLOCK-1)/NTHREADS_PER_BLOCK,NTHREADS_PER_BLOCK>>>(dev_a,dev_b,patchsize,n);


  float* dev_patches;
  hipMalloc((void**)&dev_patches,(n*n)*(patchsize*patchsize)*sizeof(float));
  for (int i=0; i<patchsize; i++)
    for (int j=0; j<patchsize; j++)
      patch<<<(size*size+NTHREADS_PER_BLOCK-1)/NTHREADS_PER_BLOCK,NTHREADS_PER_BLOCK>>>(dev_b,dev_patches,i,j,size,patchsize,n);

  float* dev_d,*dev_gauss;
  hipMalloc((void**)&dev_gauss,pow(patchsize,2)*sizeof(float));
  hipMalloc((void**)&dev_d,pow(n,4)*sizeof(float));
  hipMemcpy(dev_gauss,gauss,pow(patchsize,2)*sizeof(float),hipMemcpyHostToDevice);

  // shared memory needs to be sizeof block +
  int shared_arr=(n*n)/BLOCKS;
  int r_shared_arr=shared_arr/n+2*h_patch;
  size_t size_patches=(r_shared_arr*r_shared_arr)*sizeof(float);
    for (int i=0; i<(n*n); i++)
        d_fill<<<(n*n+NTHREADS_PER_BLOCK-1)/NTHREADS_PER_BLOCK,NTHREADS_PER_BLOCK,size_patches>>>(dev_b,dev_d,dev_patches,i,patchsize,dev_gauss,n,size_patches,filtsigma);

  hipFree(dev_patches);
  hipFree(dev_b);
  hipFree(dev_gauss);


  float* dev_out;
  hipMalloc((void**)&dev_out,n*n*sizeof(float));
  matrix_vec<<<(n*n+NTHREADS_PER_BLOCK-1)/NTHREADS_PER_BLOCK,NTHREADS_PER_BLOCK>>>(dev_d,dev_a,dev_out,n);

  hipFree(dev_a);
  hipFree(dev_d);
  hipMemcpy(temp_out,dev_out,n*n*sizeof(float),hipMemcpyDeviceToHost);
  hipFree(dev_out);
  for (int i=0; i<n; i++)
    for (int j=0; j<n; j++)
      A[j][i]=temp_out[j*n+i];

  free(temp_out);
}





int main(int argc,char* argv[]){

  int rows=64;
  int cols=64;

  float** img_Arr=(float**)malloc(rows*(sizeof(float*)));
  if (img_Arr==NULL){
    printf("\nNo memory can be allocated,exiting...\n");
    exit(-1);
  }
  for (int i=0; i<rows; i++){
    img_Arr[i]=(float*)malloc(cols*sizeof(float));
    if (img_Arr[i]==NULL){
      printf("\nNo memory can be allocated,exiting...\n");
      exit(-1);
    }
  }

  printf("Reading image...\n");
  FILE* fp;
  fp=fopen("./new_image.txt","r");
  if (fp==NULL){
    printf("\nThere is no such file,exiting...\n");
    exit(1);
  }

  float num;
  for (int i=0; i<rows; i++)
    for(int j=0; j<cols; j++){
      fscanf(fp,"%f",&num);
      if (num==EOF)
        break;
      img_Arr[i][j]=num;
    }
  fclose(fp);

  printf("\nDone Reading!\nThe rows are: %d and the columns are: %d\n",rows,cols);

  // NORMILIZING IMAGE
  printf("Normilizing Image...\n\n");
  float min=0;
  float max=0;
  for (int i=0; i<rows; i++)
    for (int j=0; j<cols; j++)
      if (min>img_Arr[i][j])
        min = img_Arr[i][j];

  for (int i=0; i<rows; i++)
    for (int j=0; j<cols; j++)
      if (max<(img_Arr[i][j]-min))
        max= img_Arr[i][j]-min;

  for (int i=0; i<rows; i++)
    for (int j=0; j<cols; j++)
        img_Arr[i][j]= (img_Arr[i][j]-min)/max;



  struct timespec ts_start,ts_end;
// image_Arr now holds the pixel values of the image

// applying gaussian noise to image
// creating a new file with name image_name+"_noised.txt"
// to store the output

  //                  PARAMETERS
  int patchsize=5;
  float noise_sigma=0.001;
  float patchsigma=5/3;
  float filtsigma=0.02;

  //*********************************************

  //             NOISE PARSING
  float** n_img_Arr=(float**)malloc(rows*sizeof(float*));
  for (int i=0; i<rows; i++)
    n_img_Arr[i]=(float*)malloc(cols*sizeof(float));

  clock_gettime(CLOCK_MONOTONIC,&ts_start);

  // aplying noise cannot be parellized due to the random nature
  // of the algorithm
  apply_Noise(rows,cols,img_Arr,n_img_Arr,noise_sigma);

  //          Manipulating File
  FILE* f_noise=fopen("noised.txt","w");
  if (f_noise==NULL){
    printf("\nCouldn't open noised image file,exiting...\n");
    exit(-1);
  }
  for (int i=0; i<rows; i++)
    for (int j=0; j<cols; j++)
      fprintf(f_noise,"%f",n_img_Arr[i][j]);
  fclose(f_noise);
  clock_gettime(CLOCK_MONOTONIC,&ts_end);
  printf("\nDone!\nTime for noise parsing: %lf \n",( (double)ts_end.tv_sec +(double)ts_end.tv_nsec*pow(10,-9)- (double)ts_start.tv_sec -(double)ts_start.tv_nsec*pow(10,-9)));


  //*******************************************

  // freeing memory from original array since it's no longer needed
  for (int i=0; i<rows; i++)
    free(img_Arr[i]);
  free(img_Arr);


  //*******************************************

  //          patch/Filter implementation


  //          Initializing residual
  float** residual=(float**)malloc(rows*sizeof(float*));
  for (int i=0; i<cols; i++)
    residual[i]=(float*)malloc(cols*sizeof(float));
  for(int i=0; i<rows; i++)
    for(int j=0; j<cols; j++)
      residual[i][j]=n_img_Arr[i][j];



  printf("\nApplying Filter...\n");
  clock_gettime(CLOCK_MONOTONIC,&ts_start);

  filter(n_img_Arr,rows,patchsize,patchsigma,filtsigma);

  //          Manipulating File
  FILE* f_filter=fopen("filtered.txt","w");
  if (f_filter==NULL){
    printf("\nCouldn't open filtered image file,exiting...\n");
    exit(-1);
  }
  for (int i=0; i<rows; i++)
    for (int j=0; j<cols; j++)
      fprintf(f_filter,"%f",n_img_Arr[i][j]);
  fclose(f_filter);
  clock_gettime(CLOCK_MONOTONIC,&ts_end);
  printf("\nDone!\nTime for filter: %lf \n",( (double)ts_end.tv_sec +(double)ts_end.tv_nsec*pow(10,-9)- (double)ts_start.tv_sec -(double)ts_start.tv_nsec*pow(10,-9)));

  //         Creating Residual
  FILE* f_ref=fopen("residual.txt","w");
  for (int i=0; i<rows; i++)
    for (int j=0; j<cols; j++)
      fprintf(f_ref,"%f",fabs(n_img_Arr[i][j]-residual[i][j]));
  fclose(f_ref);


  //*******************************************
  return 0;
}

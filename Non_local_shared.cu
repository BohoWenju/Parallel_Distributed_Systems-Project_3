/*

                      MOURELATOS ELEFTHERIOS
                            AEM 9437

    Implementing the Non_local means filter with the use of gpu
    and also taking advantage of the shared memory.
    Each block of elements of the expanded arrays is being stored
    in shared memory in order to retrieve the patches for each element
    from there.
    Instead of storing the array of pathces i store a part of B
    and retrieve each patch from there.The size of the shared memory
    is equal to the number of elements in a block of the original array
    expanded by h_patch both in rows and cols.




*/






#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <string.h>
#include <time.h>
#include <math.h>
#include <hip/hip_runtime.h>

#include <hiprand/hiprand_kernel.h>

#define NTHREADS_PER_BLOCK 256
#define BLOCKS 64

float inverse_gausian(float median,float sigma,float random_value){
  float y1=median + sqrt(-2*pow(sigma,2)*log(sqrt(2*M_PI)*sigma*random_value));
  float y2=median - sqrt(-2*pow(sigma,2)*log(sqrt(2*M_PI)*sigma*random_value));
  float x=(float)rand()/RAND_MAX;
  return (x<0.5)?y1:y2;
}

void apply_Noise(int rows,int cols,float** In,float** Out,float sigma){
  for (int i=0; i<rows; i++)
    for (int j=0; j<cols; j++){
      double z=inverse_gausian(In[i][j],sigma,(float)rand()/RAND_MAX);
      if (z > 1)
        Out[i][j]=1;
      else if (z < 0)
        Out[i][j]=0;
      else
        Out[i][j]=z;
    }
}

__global__
void d_fill(float* B,float* d,float* patches,int i,int patchsize,float* gauss,int n,float filtsigma){

  // each block has i.e. p^2 elements starting from the top row
  // therefore number of cols of specific block is n
  // and blockdim = rows*n ==> rows=(blockdim/n)
  // filling the array of patches is like a 2d stencil problem
  // passing the patches of one block in a shared memory
  int h_patch=(patchsize-1)/2;
  int size_rows=n+2*h_patch; // basically number of columns
  extern __shared__ float sh_patches[];
  int index = blockIdx.x*blockDim.x+threadIdx.x;

  // adding h_patch in order to navigate through the B array
  // basically copying a part of B array to shared memory
  int x=index/n+h_patch;
  int y=index%n+h_patch;
  int l_x=threadIdx.x/n;
  int l_y=threadIdx.x%n;
  if (index<(n*n)){
    sh_patches[(l_x+h_patch)*size_rows+(l_y+h_patch)]=B[x*size_rows+y];
    if ((l_y<h_patch)&&(l_x<h_patch)){
      sh_patches[l_x*size_rows+l_y]=B[(x-h_patch)*size_rows+(y-h_patch)];
      sh_patches[(l_x+h_patch+blockDim.x/n)*size_rows+(l_y+h_patch+n)]=B[(x+n)*size_rows+(y+n)];
    }
    else if (l_y<h_patch){
      sh_patches[(l_x+h_patch)*size_rows+l_y]=B[x*size_rows+(y-h_patch)];
      sh_patches[(l_x+h_patch)*size_rows+(l_y+h_patch+n)]=B[x*size_rows+(y+n)];
    }
    else if ((l_x<h_patch)){
      sh_patches[(l_x)*size_rows+(l_y+h_patch)]=B[(x-h_patch)*size_rows+y];
      sh_patches[(l_x+h_patch+blockDim.x/n)*size_rows+(l_y+h_patch)]=B[(x+n)*size_rows+y];
    }
  }

  __syncthreads();

  // d_filling the array by taking advantage the shared memory above
  // each element is (x+blockIdx.x),(y+blockIdx.x)
  // if i belongs to the same block as our thread then both readings
  // come from shared memory
  // else the reading of element i comes from global


  int f_index=(l_x)*n*patchsize*patchsize+(l_y)*patchsize*patchsize;
  // normilizing our data
  x=i/n;
  y=i%n;
  float sum=0;
  // in order to check whether the element comes from the same block
  if ((i/blockDim.x)==(blockIdx.x)){
    int n_i=i%blockDim.x;
    x=n_i/n;
    y=n_i%n;
    int s_index=x*n*patchsize*patchsize+y*patchsize*patchsize;
    if (index < powf(n,2)){
      if (index==n_i)
        d[index*n*n+i]=1;
      else {
        for (int j=0; j<patchsize; j++)
          for (int k=0; k<patchsize; k++)
            sum+=gauss[j]*powf(sh_patches[f_index+k*patchsize+j]-sh_patches[s_index+k*patchsize+j],2);
        d[index*n*n+i]=expf(-sum/powf(filtsigma,2));
        }
      }
    }
    if (index==i)
      d[index*n*n+i]=1;
    else{
      x=i/n;
      y=i%n;
      int s_index=x*n*patchsize*patchsize+y*patchsize*patchsize;
      for (int j=0; j<patchsize; j++)
        for (int k=0; k<patchsize; k++)
          sum+=gauss[j]*powf(sh_patches[f_index+k*patchsize+j]-patches[s_index+k*patchsize+j],2);
      d[index*n*n+i]=expf(-sum/powf(filtsigma,2));
    }
}

// get the patch of an image in points (x,y)
// with patchsize
__global__
void patch(float* B,float* patches,int i,int j,int size,int patchsize,int n){
  int index = blockIdx.x * NTHREADS_PER_BLOCK + threadIdx.x;
  int h_patch=(patchsize-1)/2;
  int x=index/size;
  int y=index%size;
    if ((x<size)&&(y<size)){
        if ((y>=h_patch)&&(y<n+h_patch)&&(x<n+h_patch)&&(x>=h_patch)){
            int patch_index=(x-h_patch)*n*patchsize*patchsize+(y-h_patch)*patchsize*patchsize+j*patchsize+i;
            patches[patch_index]=B[(x+j-h_patch)*size+(y+i-h_patch)];
    }
  }
}


// function to mirror the array (not exactly mirroring...just expanding it with 0s)
__global__
void mirroring(float* A,float* B,int patchsize,int n){
  int h_patch=(patchsize-1)/2;
  int size=n+2*h_patch;
  int index = blockIdx.x * NTHREADS_PER_BLOCK + threadIdx.x;
  int x= index/size; // row
  int y= index%size; // col
  if (index<(size*size))
    if ((x<h_patch)||(x>=n+h_patch)||(y<h_patch)||(y>=n+h_patch))
      B[index]=0;
    else
      B[index]=A[(x-h_patch)*n+(y-h_patch)];
}

__global__
void matrix_vec(float* d,float* vec,float* out,int n){
  int index= blockIdx.x*blockDim.x+threadIdx.x;
  int x=index/n;
  int y=index%n;
  int d_x=x*powf(n,3);
  int d_y=y*powf(n,2);
  float sum=0;
  float sum1=0;
  if (index<(n*n)){
    for(int i=0; i<(n*n); i++){
      sum+=d[d_x+d_y+i]*vec[i];
      sum1+=d[d_x+d_y+i];
    }
    out[index]=sum/sum1;
  }
}

void filter(float** A,int n,int patchsize,float patchsigma,float filtsigma){


  int h_patch=(patchsize-1)/2;
  int size=n+2*h_patch;

  // generating image as 1d array
  float* temp_out=(float*)malloc(n*n*sizeof(float));
  for(int i=0; i<n; i++)
    for (int j=0; j<n; j++)
      temp_out[i*n+j]=A[i][j];

  // calculate gaussian weight
  float* gauss=(float*)malloc(patchsize*patchsize*sizeof(float));
  float sum=0;
  for (int i=0; i<patchsize*patchsize; i++){
    int x=i/(patchsize);
    int y=i%(patchsize);
    x=x-((patchsize)-1)/2;
    y=y-((patchsize)-1)/2;
    gauss[i]= (1/(2.0*patchsigma))*exp(-(x*x+y*y)/(2*M_PI*patchsigma));
    sum+=(1/(2.0*patchsigma))*exp(-(x*x+y*y)/(2*M_PI*patchsigma));
  }
  for (int i=0; i<patchsize*patchsize; i++)
    gauss[i]/=sum;

  // CUDA mirroring, the allocation of variables dev is not being done
  // immediately
  float *dev_b,*dev_a;
  hipMalloc((void**)&dev_a,n*n*sizeof(float));
  hipMalloc((void**)&dev_b,size*size*sizeof(float));
  hipMemcpy(dev_a,temp_out,n*n*sizeof(float),hipMemcpyHostToDevice);
  // size*size+NTHREADS_PER_BLOCK-1/NTHREADS_PER_BLOCK because if
  // size*size is < NTHREADS_PER_BLOCK integer division will give 0
  mirroring<<<(size*size+NTHREADS_PER_BLOCK-1)/NTHREADS_PER_BLOCK,NTHREADS_PER_BLOCK>>>(dev_a,dev_b,patchsize,n);


  float* dev_patches;
  hipMalloc((void**)&dev_patches,(n*n)*(patchsize*patchsize)*sizeof(float));
  for (int i=0; i<patchsize; i++)
    for (int j=0; j<patchsize; j++)
      patch<<<(size*size+NTHREADS_PER_BLOCK-1)/NTHREADS_PER_BLOCK,NTHREADS_PER_BLOCK>>>(dev_b,dev_patches,i,j,size,patchsize,n);

  float* dev_d,*dev_gauss;
  hipMalloc((void**)&dev_gauss,pow(patchsize,2)*sizeof(float));
  hipMalloc((void**)&dev_d,pow(n,4)*sizeof(float));
  hipMemcpy(dev_gauss,gauss,pow(patchsize,2)*sizeof(float),hipMemcpyHostToDevice);

  // shared memory needs to be sizeof block +
  int shared_arr=(n*n)/BLOCKS;
  int r_shared_arr=shared_arr/n+2*h_patch;
  size_t size_patches=(r_shared_arr*r_shared_arr)*sizeof(float);
    for (int i=0; i<(n*n); i++)
        d_fill<<<(n*n+NTHREADS_PER_BLOCK-1)/NTHREADS_PER_BLOCK,NTHREADS_PER_BLOCK,size_patches>>>(dev_b,dev_d,dev_patches,i,patchsize,dev_gauss,n,filtsigma);

  hipFree(dev_patches);
  hipFree(dev_b);
  hipFree(dev_gauss);


  float* dev_out;
  hipMalloc((void**)&dev_out,n*n*sizeof(float));
  matrix_vec<<<(n*n+NTHREADS_PER_BLOCK-1)/NTHREADS_PER_BLOCK,NTHREADS_PER_BLOCK>>>(dev_d,dev_a,dev_out,n);

  hipFree(dev_a);
  hipFree(dev_d);
  hipMemcpy(temp_out,dev_out,n*n*sizeof(float),hipMemcpyDeviceToHost);
  hipFree(dev_out);
  for (int i=0; i<n; i++)
    for (int j=0; j<n; j++)
      A[j][i]=temp_out[j*n+i];

  free(temp_out);
}






int main(int argc,char* argv[]){

  //  Translating INPUT


  if (argc<=1){
    printf("\nNo image was passed as input,exiting...\n");
    exit(1);
  }
  if (argc <=2){
    printf("\nNo image dimensions (dim1xdim2) were parsed,exiting...\n");
    exit(1);
  }

  char* img_name;
  img_name=(char*)malloc(strlen(argv[1])*sizeof(char));
  if (img_name==NULL){
    printf("\nNo memory can be allocated,exiting...\n");
    exit(-1);
  }
  memcpy(img_name,argv[1],strlen(argv[1]));

  // getiing dimensions of image from input
  bool flag=true;
  char *rows_str,*cols_str;
  int size_r=0;
  int size=strlen(argv[2]);
  for (int i=0; i<strlen(argv[2]); i++){
    if (((argv[2][i]=='X')||(argv[2][i]=='x'))&&(flag)){
      rows_str=(char*)malloc((i-1)*sizeof(char));
      for (int j=0; j<i; j++)
        rows_str[j]=argv[2][j];
      flag=false;
      size_r=i+1;
    }
    if ((!flag)&&(i==(strlen(argv[2])-1))){
      cols_str=(char*)malloc((i-size_r+1)*sizeof(char));
      for (int j=size_r; j<=i; j++)
        cols_str[j-size_r]=argv[2][j];
    }
  }
  int rows=atoi(rows_str);
  int cols=atoi(cols_str);
  free(rows_str);
  free(cols_str);


  float** img_Arr=(float**)malloc(rows*(sizeof(float*)));
  if (img_Arr==NULL){
    printf("\nNo memory can be allocated,exiting...\n");
    exit(-1);
  }
  for (int i=0; i<rows; i++){
    img_Arr[i]=(float*)malloc(cols*sizeof(float));
    if (img_Arr[i]==NULL){
      printf("\nNo memory can be allocated,exiting...\n");
      exit(-1);
    }
  }

  printf("Reading image...\n");
  FILE* fp;
  fp=fopen(img_name,"r");
  if (fp==NULL){
    printf("\nThere is no such file,exiting...\n");
    exit(1);
  }

  float num;
  for (int i=0; i<rows; i++)
    for(int j=0; j<cols; j++){
      fscanf(fp,"%f",&num);
      if (num==EOF)
        break;
      img_Arr[i][j]=num;
    }
  fclose(fp);

  printf("\nDone Reading!\nThe rows are: %d and the columns are: %d\n",rows,cols);

  // NORMILIZING IMAGE
  printf("Normilizing Image...\n\n");
  float min=0;
  float max=0;
  for (int i=0; i<rows; i++)
    for (int j=0; j<cols; j++)
      if (min>img_Arr[i][j])
        min = img_Arr[i][j];

  for (int i=0; i<rows; i++)
    for (int j=0; j<cols; j++)
      if (max<(img_Arr[i][j]-min))
        max= img_Arr[i][j]-min;

  for (int i=0; i<rows; i++)
    for (int j=0; j<cols; j++)
        img_Arr[i][j]= (img_Arr[i][j]-min)/max;



  struct timespec ts_start,ts_end;
  // image_Arr now holds the pixel values of the image

  // creating names for the new files:
  // image_name+"_noised.txt"
  // image_name+_filtered.txt"
  // image_name+_residual.txt
  // to store the output

/*
  for (int i=0; i<4; i++)
    img_name[strlen(img_name)-1]='\0';

  char* n_img_name=(char*)malloc(strlen(img_name)*sizeof(char));
  if (n_img_name==NULL){
    printf("\nNo memory can be allocated,exiting...\n");
    exit(-1);
  }
  memcpy(n_img_name,img_name,strlen(img_name));
  strcat(n_img_name,"_noised.txt");

  char* img_filt=(char*)malloc(strlen(img_name)*sizeof(char));
  if (img_filt==NULL){
    printf("\nNo memory can be allocated,exiting...\n");
    exit(-1);
  }
  memcpy(img_filt,img_name,strlen(img_name));
  strcat(img_filt,"_filtered.txt");

  char* img_res=(char*)malloc(strlen(img_name)*sizeof(char));
  if (img_res==NULL){img_filt
    printf("\nNo memory can be allocated,exiting...\n");
    exit(-1);
  }
  memcpy(img_res,img_name,strlen(img_name));
  strcat(n_img_name,"_residual.txt");
*/
  free(img_name);


  //*********************************************

  //                  PARAMETERS

  int patchsize=5;
  float noise_sigma=0.01;
  float patchsigma=0.01;
  float filtsigma=1;

  //*********************************************


  //*********************************************

  //             NOISE PARSING

  float** n_img_Arr=(float**)malloc(rows*sizeof(float*));
  for (int i=0; i<rows; i++)
    n_img_Arr[i]=(float*)malloc(cols*sizeof(float));

  clock_gettime(CLOCK_MONOTONIC,&ts_start);

  // aplying noise cannot be parellized due to the random nature
  // of the algorithm
  apply_Noise(rows,cols,img_Arr,n_img_Arr,noise_sigma);

  //          Manipulating File
  FILE* f_noise=fopen("noised.txt","w");
  if (f_noise==NULL){
    printf("\nCouldn't open noised image file,exiting...\n");
    exit(-1);
  }
  for (int i=0; i<rows; i++)
    for (int j=0; j<cols; j++)
      fprintf(f_noise,"%f",n_img_Arr[i][j]);
  fclose(f_noise);
  clock_gettime(CLOCK_MONOTONIC,&ts_end);
  printf("\nDone!\nTime for noise parsing: %lf \n",( (double)ts_end.tv_sec +(double)ts_end.tv_nsec*pow(10,-9)- (double)ts_start.tv_sec -(double)ts_start.tv_nsec*pow(10,-9)));

  // freeing memory from original array since it's no longer needed
  for (int i=0; i<rows; i++)
    free(img_Arr[i]);
  free(img_Arr);

  //*******************************************




  //*******************************************

  //          Patch/Filter implementation

  //          Initializing residual
  float** residual=(float**)malloc(rows*sizeof(float*));
  for (int i=0; i<cols; i++)
    residual[i]=(float*)malloc(cols*sizeof(float));
  for(int i=0; i<rows; i++)
    for(int j=0; j<cols; j++)
      residual[i][j]=n_img_Arr[i][j];



  printf("\nApplying Filter...\n");
  clock_gettime(CLOCK_MONOTONIC,&ts_start);

  filter(n_img_Arr,rows,patchsize,patchsigma,filtsigma);

  clock_gettime(CLOCK_MONOTONIC,&ts_end);
  printf("\nDone!\nTime for filter: %lf \n",( (double)ts_end.tv_sec +(double)ts_end.tv_nsec*pow(10,-9)- (double)ts_start.tv_sec -(double)ts_start.tv_nsec*pow(10,-9)));

  //          Creating Filtered image
  FILE* f_filter=fopen("filtered.txt","w");
  if (f_filter==NULL){
    printf("\nCouldn't open filtered image file,exiting...\n");
    exit(-1);
  }
  for (int i=0; i<rows; i++)
    for (int j=0; j<cols; j++)
      fprintf(f_filter,"%f",n_img_Arr[i][j]);
  fclose(f_filter);

  //         Creating Residual image
  FILE* f_ref=fopen("residual.txt","w");
  for (int i=0; i<rows; i++)
    for (int j=0; j<cols; j++)
      fprintf(f_ref,"%f",fabs(n_img_Arr[i][j]-residual[i][j]));
  fclose(f_ref);


  //*******************************************
  return 0;
}
